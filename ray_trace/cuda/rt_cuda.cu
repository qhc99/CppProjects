#include "hip/hip_runtime.h"
#include "rtweekend.h"
#include "write_color.h"
#include "HittableList.h"
#include "Sphere.h"
#include "Camera.h"
#include "Material.h"
#include <execution>


__device__ Color ray_color(const Ray &r, const Hittable &world, int depth) {
    HitRecord rec;

    // If we've exceeded the ray bounce limit, no more light is gathered.
    if (depth <= 0) {
        return {0, 0, 0};
    }
    if (world.hit(r, 0.001, infinity, rec)) {
        Ray scattered;
        Color attenuation;
        if (rec.mat_ptr->scatter(r, rec, attenuation, scattered)) {
            return attenuation * ray_color(scattered, world, depth - 1);
        }
        return {0, 0, 0};
    }

    Vec3 unit_direction = unit_vector(r.direction());
    auto t = 0.5 * (unit_direction.y() + 1.0);
    return (1.0 - t) * Color(1.0, 1.0, 1.0) + t * Color(0.5, 0.7, 1.0);
}

__device__ HittableList random_scene() {
    HittableList world{40};

    auto ground_material = new Lambertian(Color(0.5, 0.5, 0.5));
    world.add(new Sphere(Point3(0, -1000, 0), 1000, ground_material));

    for (int a = -11; a < 11; a++) {
        for (int b = -11; b < 11; b++) {
            auto choose_mat = random_double();
            Point3 center(a + 0.9 * random_double(), 0.2, b + 0.9 * random_double());

            if ((center - Point3(4, 0.2, 0)).length() > 0.9) {
                Material *sphere_material;

                if (choose_mat < 0.8) {
                    // diffuse
                    auto albedo = Color::random() * Color::random();
                    sphere_material = new Lambertian(albedo);
                    world.add(new Sphere(center, 0.2, sphere_material));
                } else if (choose_mat < 0.95) {
                    // metal
                    auto albedo = Color::random(0.5, 1);
                    auto fuzz = random_double(0, 0.5);
                    sphere_material = new Metal(albedo, fuzz);
                    world.add(new Sphere(center, 0.2, sphere_material));
                } else {
                    // glass
                    sphere_material = new Dielectric(1.5);
                    world.add(new Sphere(center, 0.2, sphere_material));
                }
            }
        }
    }

    auto material1 = new Dielectric(1.5);
    world.add(new Sphere(Point3(0, 1, 0), 1.0, material1));

    auto material2 = new Lambertian(Color(0.4, 0.2, 0.1));
    world.add(new Sphere(Point3(-4, 1, 0), 1.0, material2));

    auto material3 = new Metal(Color(0.7, 0.6, 0.5), 0.0);
    world.add(new Sphere(Point3(4, 1, 0), 1.0, material3));

    return world;
}

constexpr auto aspect_ratio = 3.0 / 2.0;
constexpr int image_width = 300; // 1200
constexpr int image_height = static_cast<int>(image_width / aspect_ratio);
constexpr int samples_per_pixel = 100; // 500
constexpr int max_depth = 50;

__device__ void set_up() {
    auto world = random_scene();

    // Camera

    Point3 lookfrom(13, 2, 3);
    Point3 lookat(0, 0, 0);
    Vec3 vup(0, 1, 0);
    auto dist_to_focus = 10.0;
    auto aperture = 0.1;

    //Camera cam(lookfrom, lookat, vup, 20, aspect_ratio, aperture, dist_to_focus);
}

__global__ void ray_trace(HittableList *world_dev, Camera *cam_dev, Color *color_store_dev) {
    auto x = threadIdx.x + blockIdx.x * blockDim.x;
    auto y = threadIdx.y + blockIdx.y * blockDim.y;
    for (auto j = x; j < image_height; j+=gridDim.x * blockDim.x) {
        for (auto i = y; i < image_width; i+=gridDim.y*blockDim.y) {
            auto *pixel_color = new Color(0, 0, 0); // free when terminate
            for (int s = 0; s < samples_per_pixel; ++s) {
                auto u = (i + random_double()) / (image_width - 1);
                auto v = (j + random_double()) / (image_height - 1);
                Ray r = cam_dev->get_ray(u, v);
                *pixel_color += ray_color(r, *world_dev, max_depth);
            }
            color_store_dev[j * image_width + i] = *pixel_color;
        }
    }
}

int grid_dim(int desired, int block_dim, int up_limit) {
    if (desired * block_dim <= up_limit) {
        return desired;
    } else {
        return ceil(up_limit / static_cast<float>(block_dim));
    }
}

int main() {

    const int block_dim_x_y = 16;
    dim3 block_dims(block_dim_x_y, block_dim_x_y);
    dim3 grid_dims(
        grid_dim(256, block_dim_x_y, image_height),
        grid_dim(256, block_dim_x_y, image_width));

    std::cout << "P3\n" << image_width << " " << image_height << "\n255\n";

    //---------------------------------------------------
    auto **color_store = new Color *[image_height];
    for (int i = 0; i < image_height; i++) {
        color_store[i] = static_cast<Color *>(malloc(sizeof(Color) * (image_width)));
    }

    for (int j = image_height - 1; j >= 0; --j) {
        for (int i = 0; i < image_width; ++i) {
            write_color(std::cout, color_store[j][i], samples_per_pixel);
        }
    }

    std::cerr << "\nDone.\n";
}