#include "hip/hip_runtime.h"
#include "rtweekend.h"
#include "write_color.h"
#include "HittableList.h"
#include "Sphere.h"
#include "Camera.h"
#include "Material.h"
#include "utils.h"


__device__ Color ray_color(const Ray &r, const Hittable *world, int depth, hiprandState *state) {
    HitRecord rec;

    // If we've exceeded the ray bounce limit, no more light is gathered.
    if (depth <= 0) {
        return {0, 0, 0};
    }
    if (world->hit(r, 0.001, infinity, rec)) {
        Ray scattered;
        Color attenuation;
        if (rec.mat_ptr->scatter(r, rec, attenuation, scattered, state)) {
            return attenuation * ray_color(scattered, world, depth - 1, state);
        }
        return {0, 0, 0};
    }

    Vec3 unit_direction = unit_vector(r.direction());
    auto t = 0.5 * (unit_direction.y() + 1.0);
    return (1.0 - t) * Color(1.0, 1.0, 1.0) + t * Color(0.5, 0.7, 1.0);
}

constexpr int cache_size = 1000;

__device__ void random_scene(
    HittableList *world_dev,
    Sphere *sphere_cache,
    Hittable **hittable_cache,
    Lambertian *lambertian_cache,
    Metal *metal_cache,
    Dielectric *dielectric_cache,
    hiprandState *state
) {

    new(world_dev) HittableList(cache_size, hittable_cache);
    int sphere_idx = 0, lam_idx = 0, metal_idx = 0, dielec_idx = 0;

    new(&lambertian_cache[lam_idx])Lambertian(Color(0.5, 0.5, 0.5));
    auto ground_material = &lambertian_cache[lam_idx];
    lam_idx++;
    new(&sphere_cache[sphere_idx])Sphere(Point3(0, -1000, 0), 1000, ground_material);
    world_dev->add(&sphere_cache[sphere_idx]);
    sphere_idx++;

    // should consider cache size
    for (int a = -11; a < 11; a++) {
        for (int b = -11; b < 11; b++) {
            auto choose_mat = random_double(state);
            Point3 center(a + 0.9 * random_double(state), 0.2, b + 0.9 * random_double(state));

            if ((center - Point3(4, 0.2, 0)).length() > 0.9) {
                Material *sphere_material;

                if (choose_mat < 0.8) {
                    // diffuse
                    auto albedo = Color::random(state) * Color::random(state);

                    new(&lambertian_cache[lam_idx])Lambertian(albedo);
                    sphere_material = &lambertian_cache[lam_idx];
                    lam_idx++;
                    new(&sphere_cache[sphere_idx])Sphere(center, 0.2, sphere_material);
                    world_dev->add(&sphere_cache[sphere_idx]);
                    sphere_idx++;
                } else if (choose_mat < 0.95) {
                    // metal
                    auto albedo = Color::random(0.5, 1, state);
                    auto fuzz = random_double(0, 0.5, state);
                    new(&metal_cache[metal_idx])Metal(albedo, fuzz);
                    sphere_material = &metal_cache[metal_idx];
                    metal_idx++;
                    new(&sphere_cache[sphere_idx])Sphere(center, 0.2, sphere_material);
                    world_dev->add(&sphere_cache[sphere_idx]);
                    sphere_idx++;
                } else {
                    // glass
                    new(&dielectric_cache[dielec_idx]) Dielectric(1.5);
                    sphere_material = &dielectric_cache[dielec_idx];
                    dielec_idx++;
                    new(&sphere_cache[sphere_idx])Sphere(center, 0.2, sphere_material);
                    world_dev->add(&sphere_cache[sphere_idx]);
                    sphere_idx++;
                }
            }
        }
    }
    new(&dielectric_cache[dielec_idx])Dielectric(1.5);
    auto material1 = &dielectric_cache[dielec_idx];
    new(&sphere_cache[sphere_idx])Sphere(Point3(0, 1, 0), 1.0, material1);
    world_dev->add(&sphere_cache[sphere_idx]);
    sphere_idx++;

    new(&lambertian_cache[lam_idx])Lambertian(Color(0.4, 0.2, 0.1));
    auto material2 = &lambertian_cache[lam_idx];
    new(&sphere_cache[sphere_idx]) Sphere(Point3(-4, 1, 0), 1.0, material2);
    world_dev->add(&sphere_cache[sphere_idx]);
    sphere_idx++;

    new(&metal_cache[metal_idx])Metal(Color(0.7, 0.6, 0.5), 0.0);
    auto material3 = &metal_cache[metal_idx];
    new(&sphere_cache[sphere_idx])Sphere(Point3(4, 1, 0), 1.0, material3);
    world_dev->add(&sphere_cache[sphere_idx]);
}

constexpr auto aspect_ratio = 3.0 / 2.0;
constexpr int image_width = 1200; // 1200
constexpr int image_height = static_cast<int>(image_width / aspect_ratio);
constexpr int samples_per_pixel = 500; // 500
constexpr int max_depth = 50;

__global__ void set_up(
    HittableList *world_dev,
    Sphere *sphere_cache,
    Hittable **hittable_cache,
    Lambertian *lambertian_cache,
    Metal *metal_cache,
    Dielectric *dielectric_cache,
    Camera *cam_dev,
    hiprandState *state
) {

    // init rand state
    hiprand_init(clock(), 0, 0, state);
    // World
    random_scene(
        world_dev,
        sphere_cache,
        hittable_cache,
        lambertian_cache,
        metal_cache,
        dielectric_cache,
        state);

    // Camera
    Point3 look_from(13, 2, 3);
    Point3 look_at(0, 0, 0);
    Vec3 vup(0, 1, 0);
    auto dist_to_focus = 10.0;
    auto aperture = 0.1;
    new(cam_dev) Camera(look_from, look_at, vup, 20, aspect_ratio, aperture, dist_to_focus);


}

__global__ void ray_trace(HittableList *world_dev, Camera *cam_dev, Color *color_store_dev, hiprandState *state) {
    auto x = threadIdx.x + blockIdx.x * blockDim.x;
    auto y = threadIdx.y + blockIdx.y * blockDim.y;
    for (auto j = x; j < image_height; j += gridDim.x * blockDim.x) {
        for (auto i = y; i < image_width; i += gridDim.y * blockDim.y) {
            auto pixel_color = Color(0, 0, 0); // free when terminate
            for (int s = 0; s < samples_per_pixel; ++s) {
                auto u = (i + random_double(state)) / (image_width - 1);
                auto v = (j + random_double(state)) / (image_height - 1);
                Ray r = cam_dev->get_ray(u, v, state);
                pixel_color += ray_color(r, world_dev, max_depth, state);
            }
            color_store_dev[j * image_width + i] = pixel_color;
        }
    }
}

int grid_dim(int desired, int block_dim, int up_limit) {
    if (desired * block_dim <= up_limit) {
        return desired;
    } else {
        return ceil(static_cast<float>(up_limit) / static_cast<float>(block_dim));
    }
}

int main() {

    const int block_dim_x_y = 16;
    dim3 block_dims(block_dim_x_y, block_dim_x_y);
    dim3 grid_dims(
        grid_dim(256, block_dim_x_y, image_height),
        grid_dim(256, block_dim_x_y, image_width));

    std::cout << "P3\n" << image_width << " " << image_height << "\n255\n";

    // var
    auto *color_store = static_cast<Color *>(malloc(sizeof(Color) * image_width * image_height));
    Color *color_store_dev{nullptr};
    HANDLE_ERROR(hipMalloc(&color_store_dev, sizeof(Color) * image_width * image_height));
    HittableList *world_dev{nullptr};
    HANDLE_ERROR(hipMalloc(&world_dev, sizeof(HittableList)));
    Camera *cam_dev{nullptr};
    HANDLE_ERROR(hipMalloc(&cam_dev, sizeof(Camera)));
    hiprandState *rnd_state_dev;
    HANDLE_ERROR(hipMalloc(&rnd_state_dev, sizeof(hiprandState)));

    // cache
    Sphere *sphere_cache_dev{nullptr};
    HANDLE_ERROR(hipMalloc(&sphere_cache_dev, sizeof(Sphere) * cache_size));
    Hittable **hittable_cache_dev{nullptr};
    HANDLE_ERROR(hipMalloc(&hittable_cache_dev, sizeof(Hittable *) * cache_size));
    Lambertian *lambertian_cache_dev{nullptr};
    HANDLE_ERROR(hipMalloc(&lambertian_cache_dev, sizeof(Lambertian) * cache_size));
    Metal *metal_cache_dev{nullptr};
    HANDLE_ERROR(hipMalloc(&metal_cache_dev, sizeof(Metal) * cache_size));
    Dielectric *dielectric_cache_dev{nullptr};
    HANDLE_ERROR(hipMalloc(&dielectric_cache_dev, sizeof(Dielectric) * cache_size));

    // set up
    set_up<<<1, 1>>>(
        world_dev,
        sphere_cache_dev,
        hittable_cache_dev,
        lambertian_cache_dev,
        metal_cache_dev,
        dielectric_cache_dev,
        cam_dev,
        rnd_state_dev);

    hipDeviceSynchronize();
    // metric var
    hipEvent_t start, stop;
    HANDLE_ERROR(hipEventCreate(&start));
    HANDLE_ERROR(hipEventCreate(&stop));
    HANDLE_ERROR(hipEventRecord(start));


    // kernel
    ray_trace<<<grid_dims, block_dims>>>(world_dev, cam_dev, color_store_dev, rnd_state_dev);

    HANDLE_ERROR(hipEventRecord(stop));
    HANDLE_ERROR(hipEventSynchronize(stop));
    float elapsedTime;
    HANDLE_ERROR(hipEventElapsedTime(&elapsedTime,
                                      start, stop));
    printf("Time to generate:  %3.1f ms\n", elapsedTime);

    // output image
    hipMemcpy(color_store, color_store_dev, sizeof(Color) * image_height * image_width, hipMemcpyDeviceToHost);
    for (int j = image_height - 1; j >= 0; --j) {
        for (int i = 0; i < image_width; ++i) {
            write_color(std::cout, color_store[j * image_width + i], samples_per_pixel);
        }
    }
    std::cerr << "\nDone.\n";

    HANDLE_ERROR(hipFree(world_dev));
    HANDLE_ERROR(hipFree(cam_dev));
    HANDLE_ERROR(hipFree(color_store_dev));
    HANDLE_ERROR(hipFree(rnd_state_dev));
    HANDLE_ERROR(hipFree(sphere_cache_dev));
    HANDLE_ERROR(hipFree(hittable_cache_dev));
    HANDLE_ERROR(hipFree(lambertian_cache_dev));
    HANDLE_ERROR(hipFree(metal_cache_dev));
    HANDLE_ERROR(hipFree(dielectric_cache_dev));
    free(color_store);
}