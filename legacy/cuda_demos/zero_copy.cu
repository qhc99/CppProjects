//
// Created by QC on 2022-12-23.
//

#include <hip/hip_runtime.h>
#include <cstdio>
#define imin(a,b) (a<b?a:b)
const int N = 33 * 1024 * 1024;
const int threadsPerBlock = 256;
const int blocksPerGrid =
        imin( 32, (N+threadsPerBlock-1) / threadsPerBlock );


__global__ void dot( int size, float *a, float *b, float *c ) {
    __shared__ float cache[threadsPerBlock];
    int tid = threadIdx.x + blockIdx.x * blockDim.x;
    int cacheIndex = threadIdx.x;
    float temp = 0;
    while (tid < size) {
        temp += a[tid] * b[tid];
        tid += blockDim.x * gridDim.x;
    }
// set the cache values
    cache[cacheIndex] = temp;
// synchronize threads in this block
    __syncthreads();
// for reductions, threadsPerBlock must be a power of 2
// because of the following code
    int i = blockDim.x/2;
    while (i != 0) {
        if (cacheIndex < i)
            cache[cacheIndex] += cache[cacheIndex + i];
        __syncthreads();
        i /= 2;
    }
    if (cacheIndex == 0)
        c[blockIdx.x] = cache[0];
}

float malloc_test( int size ) {
    hipEvent_t start, stop;
    float *a, *b, c, *partial_c;
    float *dev_a, *dev_b, *dev_partial_c;
    float elapsedTime;
    ( hipEventCreate( &start ) );
    ( hipEventCreate( &stop ) );
// allocate memory on the CPU side
    a = (float*)malloc( size*sizeof(float) );
    b = (float*)malloc( size*sizeof(float) );
    partial_c = (float*)malloc( blocksPerGrid*sizeof(float) );
// allocate the memory on the GPU
    ( hipMalloc( (void**)&dev_a,
                              size*sizeof(float) ) );
    ( hipMalloc( (void**)&dev_b,
                              size*sizeof(float) ) );
    ( hipMalloc( (void**)&dev_partial_c,
                              blocksPerGrid*sizeof(float) ) );
// fill in the host memory with data
    for (int i=0; i<size; i++) {
        a[i] = i;
        b[i] = i*2;
    }

    ( hipEventRecord( start, 0 ) );
// copy the arrays 'a' and 'b' to the GPU
    ( hipMemcpy( dev_a, a, size*sizeof(float),
                              hipMemcpyHostToDevice ) );
    ( hipMemcpy( dev_b, b, size*sizeof(float),
                              hipMemcpyHostToDevice ) );
    dot<<<blocksPerGrid,threadsPerBlock>>>( size, dev_a, dev_b,
            dev_partial_c );
// copy the array 'c' back from the GPU to the CPU
    ( hipMemcpy( partial_c, dev_partial_c,
                              blocksPerGrid*sizeof(float),
                              hipMemcpyDeviceToHost ) );

                              ( hipEventRecord( stop, 0 ) );
    ( hipEventSynchronize( stop ) );
    ( hipEventElapsedTime( &elapsedTime,
                                        start, stop ) );

    // finish up on the CPU side
    c = 0;
    for (int i=0; i<blocksPerGrid; i++) {
        c += partial_c[i];
    }

    ( hipFree( dev_a ) );
    ( hipFree( dev_b ) );
    ( hipFree( dev_partial_c ) );
// free memory on the CPU side
    free( a );
    free( b );
    free( partial_c );
// free events
    ( hipEventDestroy( start ) );
    ( hipEventDestroy( stop ) );
    printf( "Value calculated: %f\n", c );
    return elapsedTime;
}

float cuda_host_alloc_test( int size ) {
    hipEvent_t start, stop;
    float *a, *b, c, *partial_c;
    float *dev_a, *dev_b, *dev_partial_c;
    float elapsedTime;
    ( hipEventCreate( &start ) );
    ( hipEventCreate( &stop ) );
// allocate the memory on the CPU
    ( hipHostAlloc( (void**)&a,
                                 size*sizeof(float),
                                 hipHostMallocWriteCombined |
                                 hipHostMallocMapped ) );
    ( hipHostAlloc( (void**)&b,
                                 size*sizeof(float),
                                 hipHostMallocWriteCombined |
                                 hipHostMallocMapped ) );
    ( hipHostAlloc( (void**)&partial_c,
                                 blocksPerGrid*sizeof(float),
                                 hipHostMallocMapped ) );
// fill in the host memory with data
    for (int i=0; i<size; i++) {
        a[i] = i;
        b[i] = i*2;
    }
    ( hipHostGetDevicePointer( &dev_a, a, 0 ) );
    ( hipHostGetDevicePointer( &dev_b, b, 0 ) );
    ( hipHostGetDevicePointer( &dev_partial_c,
                                            partial_c, 0 ) );
    ( hipEventRecord( start, 0 ) );
    dot<<<blocksPerGrid,threadsPerBlock>>>( size, dev_a, dev_b,
                                            dev_partial_c );
    ( hipDeviceSynchronize() );
    ( hipEventRecord( stop, 0 ) );
    ( hipEventSynchronize( stop ) );
    ( hipEventElapsedTime( &elapsedTime,
                                        start, stop ) );
// finish up on the CPU side
    c = 0;
    for (int i=0; i<blocksPerGrid; i++) {
        c += partial_c[i];
    }
    ( hipHostFree( a ) );
    ( hipHostFree( b ) );
    ( hipHostFree( partial_c ) );
// free events
    ( hipEventDestroy( start ) );
    ( hipEventDestroy( stop ) );
    printf( "Value calculated: %f\n", c );
    return elapsedTime;
}

int main( void ) {
    hipDeviceProp_t prop;
    int whichDevice;
    ( hipGetDevice( &whichDevice ) );
    ( hipGetDeviceProperties( &prop, whichDevice ) );
    printf("integrated: %d\n", prop.integrated);
    if (prop.canMapHostMemory != 1) {
        printf( "Device cannot map memory.\n" );
        return 0;
    }
    hipSetDeviceFlags( hipDeviceMapHost );
    float elapsedTime = malloc_test( N );
    printf( "Time using cudaMalloc: %3.1f ms\n",
            elapsedTime );
    elapsedTime = cuda_host_alloc_test( N );
    printf( "Time using cudaHostAlloc: %3.1f ms\n",
            elapsedTime );
}